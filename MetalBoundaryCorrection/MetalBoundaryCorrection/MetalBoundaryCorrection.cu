#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <string>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <omp.h>
#include "gaussianFilter.h"
#include "hip/hip_runtime_api.h"
#include "ImageIO.h"
#include "MetalBoundaryCorrection.h"

//using namespace std;

//CUDA function prototypes
static __global__ void d_growcuts(int *d_label, int *d_newLabel, size_t d_labelPitch, float *d_strength, float *d_newStrength, size_t d_strengthPitch,
	float *d_image, size_t d_imagePitch, int *d_converged, int ncols, int nrows);
static __global__ void d_growcuts_Checker(int *d_labels, size_t d_labelPitch, float *d_strength, size_t d_strengthPitch,
	float *d_image, size_t d_imagePitch, int *d_converged, int ncols, int nrows, int d);
int iDivUp(int a, int b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}


////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::initialize()
{
	m_labels = NULL;
	m_newLabels = NULL;
	m_strength = NULL;
	m_newStrength = NULL;

	//cuda
	d_labels = NULL;
	d_newLabels = NULL;
	d_strength = NULL;
	d_newStrength = NULL;

	m_nDilate = 0;
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::dealloc()
{

	if (m_labels)
	{
		delete[] m_labels[0];
		delete[] m_labels;
	}

	if (m_newLabels)
	{
		delete[] m_newLabels[0];
		delete[] m_newLabels;
	}
	if (m_strength)
	{
		delete[] m_strength[0];
		delete[] m_strength;
	}

	if (m_newLabels)
	{
		delete[] m_newStrength[0];
		delete[] m_newStrength;
	}

	if (m_bUsingGPU)
		cleanUpCuda();
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
MetalBoundaryCorrection::MetalBoundaryCorrection()
{
	initialize();
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
MetalBoundaryCorrection::~MetalBoundaryCorrection()
{
	dealloc();
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::setup(int rows, int cols, bool bUsingGPU)
{
	m_ncols = cols;
	m_nrows = rows;

	m_bUsingGPU = bUsingGPU;

	if (m_bUsingGPU)
		initializeGPU();
	//else
		initializeCPU();
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::initializeCPU()
{
	int i;


	//allocate memory
	m_labels = new int*[m_nrows];
	m_labels[0] = new int[m_nrows*m_ncols];
	for (i = 1; i < m_nrows; i++)
	{
		m_labels[i] = m_labels[i - 1] + m_ncols;
	}

	m_newLabels = new int*[m_nrows];
	m_newLabels[0] = new int[m_nrows*m_ncols];
	for (i = 1; i < m_nrows; i++)
	{
		m_newLabels[i] = m_newLabels[i - 1] + m_ncols;
	}

	m_strength = new float*[m_nrows];
	m_strength[0] = new float[m_nrows*m_ncols];
	for (i = 1; i < m_nrows; i++)
	{
		m_strength[i] = m_strength[i - 1] + m_ncols;
	}

	m_newStrength = new float*[m_nrows];
	m_newStrength[0] = new float[m_nrows*m_ncols];
	for (i = 1; i < m_nrows; i++)
	{
		m_newStrength[i] = m_newStrength[i - 1] + m_ncols;
	}
}


////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::initializeGPU()
{
	//hipSetDevice( cutGetMaxGflopsDeviceId() );
	hipSetDevice(0);

	int blocksInX, blocksInY;

	//dimension to configure the computation grids
	m_BlockSize = dim3(16, 16, 1);
	//m_BlockSize = dim3(8, 8, 1);
	blocksInX = iDivUp(m_ncols, m_BlockSize.x);
	blocksInY = iDivUp(m_nrows, m_BlockSize.y);
	m_GridSize = dim3(blocksInX, blocksInY);


	//allocate pitch linear memory for projection image
	hipMallocPitch((void **)&d_labels, &d_labelPitch, m_ncols*sizeof(int), m_nrows);
	hipMallocPitch((void **)&d_newLabels, &d_labelPitch, m_ncols*sizeof(int), m_nrows);

	hipMallocPitch((void **)&d_strength, &d_strengthPitch, m_ncols*sizeof(float), m_nrows);
	hipMallocPitch((void **)&d_newStrength, &d_strengthPitch, m_ncols*sizeof(float), m_nrows);

	hipMallocPitch((void **)&d_image, &d_imagePitch, m_ncols*sizeof(float), m_nrows);

	//hipMallocPitch((void **)&d_mask, &d_maskPitch, m_ncols*sizeof(int), m_nrows);

	hipMalloc((void **)&d_converged, sizeof(int));

	//Test pushing and pulling d_converged on and off of the device
	//int converged = 1;
	//printf("%d\n", converged);

	//hipMemcpy((void *)d_converged, &converged, sizeof(int), hipMemcpyHostToDevice);

	//converged = 0;
	//printf("%d\n", converged);

	//hipMemcpy((void *)&converged, d_converged, sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d\n", converged);
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::FindBoundary(float **image, float **mask)
{
	m_mask = mask;
	m_image = image;

	//test();

	setInitalLabelsAndStrength();

	//test();

	if (!m_bUsingGPU)
		growcuts();
	else
		//growcutsGPU()
		growcutsGPU_Checker();

	if (m_nDilate > 0)
		dilateMask(m_nDilate);
}



////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::setInitalLabelsAndStrength()
{
	int i, j;

	m_left = m_ncols;
	m_right = 0;
	m_top = m_nrows;
	m_bottom = 0;

	float **maskBlur = new float*[m_nrows];
	maskBlur[0] = new float[m_nrows*m_ncols];
	for (i = 1; i < m_nrows; i++)
	{
		maskBlur[i] = maskBlur[i - 1] + m_ncols;
	}

	memcpy(maskBlur[0], m_mask[0], sizeof(float)*m_nrows*m_ncols);

	GaussianFilter myFilter;
	myFilter.setSigmaX(1.0f);
	myFilter.setSigmaY(1.0f);
	myFilter.setSigmaZ(0.0f);
	myFilter.initializeFilter();
	myFilter.filter(maskBlur, m_nrows, m_ncols);

	#pragma omp parallel for private(j)
	for (i = 1; i < m_nrows-1; i++)
	{
		for ( j = 1; j < m_ncols-1; j++) 
		{
			if (m_mask[i][j] == 0)
			{
				if ( m_mask[i][j - 1] != 0 || m_mask[i][j + 1] != 0 || m_mask[i - 1][j] != 0 || m_mask[i + 1][j] != 0) 
				{
					m_labels[i][j] = -1;       
					m_strength[i][j] = 1.0f;

					if (j < m_left)
						m_left = j;
					if (j > m_right)
						m_right = j;

					if (i < m_top)
						m_top = i;

					if (i > m_bottom)
						m_bottom = i;
				}
				else
				{
					m_labels[i][j] = -999;
					m_strength[i][j] = 1.0f;
				}
			}
			else if(m_mask[i][j] == 1)
			{
				if (m_mask[i][j - 1] != 1 || m_mask[i][j + 1] != 1 || m_mask[i - 1][j] != 1 || m_mask[i + 1][j] != 1)
				{
					m_labels[i][j] = 1;
					m_strength[i][j] = 1.0f;
				}
				else
				{
					m_labels[i][j] = 999;
					m_strength[i][j] = 1.0f;
				}
			}
			else
			{
				m_labels[i][j] = 0;
				m_strength[i][j] = 0.0f;
				//if (m_mask[i][j] < 0.5f)
				//{
				//	m_labels[i][j] = -1;
				//	m_strength[i][j] = 0.9f;//1.0f
				//}
				//else
				//{
				//	m_labels[i][j] = 0;
				//	m_strength[i][j] = 0.0f;
				//}
			}

			//add internal background seed points
			if ((maskBlur[i][j] - m_mask[i][j]) >= 0.03f)
			{
				m_labels[i][j] = -1;
				m_strength[i][j] = 1.0f;
			}
		}
	}

	delete[] maskBlur[0];
	delete[] maskBlur;
}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::dilateMask(int nDilate)
{
	int i, j;

	for(int k = 0;  k < nDilate; k++)
	{
		memcpy(m_strength[0], m_mask[0], sizeof(float)*m_nrows*m_ncols);

		for (i = m_top-1; i < m_bottom+1; i++)
		{
			for (j = m_left-1; j < m_right+1; j++)
			{
				if (m_mask[i][j] == 0)
				{
					if ( m_mask[i][j - 1] != 0 || m_mask[i][j + 1] != 0 || m_mask[i - 1][j] != 0 || m_mask[i + 1][j] != 0) 
					{
						m_strength[i][j] = 1.0f;
					}
				}
			}
		}

		memcpy(m_mask[0], m_strength[0], sizeof(float)*m_nrows*m_ncols);
	}

}


////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::growcuts()
{

	int i, j,  m;
	int ii, jj;
	//int Nx[] = { -1, 1,  0, 0, -1, -1, 1,  1 }; //8-neighbors
	//int Ny[] = {  0, 0, -1, 1,  1, -1, 1, -1 };
	//int nNeighbors = 8;
	int Nx[] = { -1, 1,  0, 0 }; //4-neighbors
	int Ny[] = {  0, 0, -1, 1 };
	int nNeighbors = 4;
	float del;
	float C, g;
	float maxC = 7000.0f;
	int converged;
	int MAX_ITS = 100;
	int its = 0;
	int cnt;
	float max = 0.0f;

	memcpy(m_newLabels[0], m_labels[0], sizeof(int)*m_nrows*m_ncols);
	memcpy(m_newStrength[0], m_strength[0], sizeof(float)*m_nrows*m_ncols);

	converged = 0;
	while (!converged)
	{
		its++;
		converged = 1; 

		cnt = 0;
		#pragma omp parallel for private(j)
		//for every pixel p
		for (i = m_top-1; i < m_bottom+1; i++)
		{
			for (j = m_left-1; j < m_right+1; j++)
			{
				//these pixels are frozen
				if (m_labels[i][j] == 999 || m_labels[i][j] == -999)
					continue;


				//for every neighbor q
				for (m = 0; m < nNeighbors; m++)
				{

					ii = i + Ny[m];
					jj = j + Nx[m];

					del = m_image[i][j] - m_image[ii][jj];
					C = sqrt(del*del);

					g = 1 - (C / maxC); //attack force

					if (C > max)
						max = C;

					g *= m_strength[ii][jj];

					if (g > m_strength[i][j]) //attack succeeds
					{
						m_newStrength[i][j] = g;
						m_newLabels[i][j] = m_labels[ii][jj];
						converged = 0; // keep iterating
						cnt++;
					}

				}

			}
		}

		//copy prev result
		memcpy(m_labels[0], m_newLabels[0], sizeof(int)*m_nrows*m_ncols);
		memcpy(m_strength[0], m_newStrength[0], sizeof(float)*m_nrows*m_ncols);

		//check cnt
		if (its == MAX_ITS)
			break;

		//printf("count %d\n",cnt);
	}

	//generate new mask
	for (i = 0; i < m_nrows; i++)
	{
		for (j = 0; j < m_ncols; j++)
		{
			if (m_newLabels[i][j] == 0)
				int mm = 0;
			if (m_newLabels[i][j] == -1 || m_newLabels[i][j] == 0 || m_newLabels[i][j] == -999)
				m_mask[i][j] = 0.0f;
			else
				m_mask[i][j] = 1.0f;
		}
	}

	printf("total iterations=%d\n",its);
	printf("C %f\n", max);



}
////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::growcutsGPU()
{

	int i, j;
	int converged;
	int MAX_ITS = 100;
	int its = 0;
	//int cnt;
	float max = 0.0f;
	hipError_t cudaStatus;

	//copy image to device
	cudaStatus = hipMemcpy2D((void *)d_image, d_imagePitch, (void *)&m_image[0][0], m_ncols*sizeof(float), m_ncols*sizeof(float), m_nrows, hipMemcpyHostToDevice);

	//copy initial labels to device
	cudaStatus = hipMemcpy2D((void *)d_labels, d_labelPitch, (void *)&m_labels[0][0], m_ncols*sizeof(int), m_ncols*sizeof(int), m_nrows, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy2D((void *)d_newLabels, d_labelPitch, (void *)d_labels, d_labelPitch, m_ncols*sizeof(int), m_nrows, hipMemcpyDeviceToDevice);
	cudaStatus = hipMemcpy2D((void *)d_newLabels, d_labelPitch, (void *)d_labels, d_labelPitch, d_labelPitch, m_nrows, hipMemcpyDeviceToDevice);

	//copy initial strength to device
	cudaStatus = hipMemcpy2D((void *)d_strength, d_strengthPitch, (void *)&m_strength[0][0], m_ncols*sizeof(float), m_ncols*sizeof(float), m_nrows, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy2D((void *)d_newStrength, d_strengthPitch, (void *)d_strength, d_strengthPitch, m_ncols*sizeof(float), m_nrows, hipMemcpyDeviceToDevice);
	cudaStatus = hipMemcpy2D((void *)d_newStrength, d_strengthPitch, (void *)d_strength, d_strengthPitch, d_strengthPitch, m_nrows, hipMemcpyDeviceToDevice);

	converged = 0;
	while (!converged)
	{
		its++;

		converged = 1;
		hipMemcpy((void *)d_converged, &converged, sizeof(int), hipMemcpyHostToDevice);

		d_growcuts <<<m_GridSize, m_BlockSize >>>(d_labels, d_newLabels, d_labelPitch / sizeof(int), d_strength, d_newStrength, d_strengthPitch / sizeof(float),
			d_image, d_imagePitch / sizeof(float), d_converged, m_ncols, m_nrows);

		hipDeviceSynchronize();
		getLastCudaError("d_growcuts kernel failed");

		//copy prev result
		//cudaStatus = hipMemcpy2D((void *)d_labels, d_labelPitch, (void *)d_newLabels, d_labelPitch, m_ncols*sizeof(int), m_nrows, hipMemcpyDeviceToDevice);
		//cudaStatus = hipMemcpy2D((void *)d_strength, d_strengthPitch, (void *)d_newStrength, d_strengthPitch, m_ncols*sizeof(float), m_nrows, hipMemcpyDeviceToDevice);

		cudaStatus = hipMemcpy2D((void *)d_labels, d_labelPitch, (void *)d_newLabels, d_labelPitch, d_labelPitch, m_nrows, hipMemcpyDeviceToDevice);
		cudaStatus = hipMemcpy2D((void *)d_strength, d_strengthPitch, (void *)d_newStrength, d_strengthPitch, d_strengthPitch, m_nrows, hipMemcpyDeviceToDevice);

		hipMemcpy((void *)&converged, d_converged, sizeof(int), hipMemcpyDeviceToHost);

		//printf("converged %d\n", converged);
		//check cnt
		if (its == MAX_ITS)
			break;

		//printf("count %d\n",cnt);
	}

	//copy label from device to host
	cudaStatus = hipMemcpy2D((void *)&m_newLabels[0][0], m_ncols*sizeof(int), (void *)d_newLabels, d_labelPitch, m_ncols*sizeof(int), m_nrows, hipMemcpyDeviceToHost);

	//generate new mask
	for (i = 0; i < m_nrows; i++)
	{
		for (j = 0; j < m_ncols; j++)
		{
			if (m_newLabels[i][j] == 0)
				int mm = 0;
			if (m_newLabels[i][j] == -1 || m_newLabels[i][j] == 0 || m_newLabels[i][j] == -999)
				m_mask[i][j] = 0.0f;
			else
				m_mask[i][j] = 1.0f;
		}
	}

	printf("total iterations=%d\n", its);
	printf("C %f\n", max);



}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::growcutsGPU_Checker()
{

	int i, j;
	int converged;
	int MAX_ITS = 100;
	int its = 0;
	//int cnt;
	float max = 0.0f;
	hipError_t cudaStatus;

	//copy image 
	cudaStatus = hipMemcpy2D((void *)d_image, d_imagePitch, (void *)&m_image[0][0], m_ncols*sizeof(float), m_ncols*sizeof(float), m_nrows, hipMemcpyHostToDevice);

	//copy initial labels to device
	cudaStatus = hipMemcpy2D((void *)d_labels, d_labelPitch, (void *)&m_labels[0][0], m_ncols*sizeof(int), m_ncols*sizeof(int), m_nrows, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy2D((void *)d_newLabels, d_labelPitch, (void *)d_labels, d_labelPitch, d_labelPitch, m_nrows, hipMemcpyDeviceToDevice);

	//copy initial strength to device
	cudaStatus = hipMemcpy2D((void *)d_strength, d_strengthPitch, (void *)&m_strength[0][0], m_ncols*sizeof(float), m_ncols*sizeof(float), m_nrows, hipMemcpyHostToDevice);
	//cudaStatus = hipMemcpy2D((void *)d_newStrength, d_strengthPitch, (void *)d_strength, d_strengthPitch, d_strengthPitch, m_nrows, hipMemcpyDeviceToDevice);

	converged = 0;
	while (!converged)
	{
		its++;

		converged = 1;
		hipMemcpy((void *)d_converged, &converged, sizeof(int), hipMemcpyHostToDevice);

		d_growcuts_Checker << <m_GridSize, m_BlockSize >> >(d_labels, d_labelPitch / sizeof(int), d_strength, d_strengthPitch / sizeof(float),
			d_image, d_imagePitch / sizeof(float), d_converged, m_ncols, m_nrows, 0);
		hipDeviceSynchronize();

		d_growcuts_Checker << <m_GridSize, m_BlockSize >> >(d_labels, d_labelPitch / sizeof(int), d_strength, d_strengthPitch / sizeof(float),
			d_image, d_imagePitch / sizeof(float), d_converged, m_ncols, m_nrows, 1);
		hipDeviceSynchronize();

		hipMemcpy((void *)&converged, d_converged, sizeof(int), hipMemcpyDeviceToHost);

		//printf("converged %d\n", converged);
		//check cnt
		if (its == MAX_ITS)
			break;

		//printf("count %d\n",cnt);
	}

	//copy label from device to host
	cudaStatus = hipMemcpy2D((void *)&m_labels[0][0], m_ncols*sizeof(int), (void *)d_labels, d_labelPitch, m_ncols*sizeof(int), m_nrows, hipMemcpyDeviceToHost);

	//generate new mask
	for (i = 0; i < m_nrows; i++)
	{
		for (j = 0; j < m_ncols; j++)
		{
			if (m_labels[i][j] == 0)
				int mm = 0;
			if (m_labels[i][j] == -1 || m_labels[i][j] == 0 || m_labels[i][j] == -999)
				m_mask[i][j] = 0.0f;
			else
				m_mask[i][j] = 1.0f;
		}
	}
	printf("total iterations=%d\n", its);
	printf("C %f\n", max);



}

////////////////////////////////////////////////////////////////////////
//
////////////////////////////////////////////////////////////////////////
void MetalBoundaryCorrection::cleanUpCuda()
{
		hipFree(d_labels);
		hipFree(d_newLabels);
		hipFree(d_newStrength);
		hipFree(d_strength);
}


////////////////////////////////////////////////////////////////////////
//gpu implementation of growcuts
////////////////////////////////////////////////////////////////////////
static __global__ void d_growcuts(int *d_labels, int *d_newLabels, size_t d_labelPitch, float *d_strength, float *d_newStrength, size_t d_strengthPitch,
	float *d_image, size_t d_imagePitch, int *d_converged, int ncols, int nrows)

{

	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;

	//if (j >= ncols || i >= nrows)
	//	return;

	if (j < 1 || i < 1 || j >= ncols-1 || i >= nrows-1)
		return;

	if (d_labels[i * d_labelPitch + j] == 999 || d_labels[i* d_labelPitch + j] == -999)
		return;       //pixel is frozen

	float del, g, C;
	float maxC = 7000.0f;
	int ii, jj;
	int Nx[] = { -1, 1, 0, 0 }; //4-neighbors
	int Ny[] = { 0, 0, -1, 1 };
	int nNeighbors = 4;

	//for every neighbor q
	for (int m = 0; m < nNeighbors; m++)
	{

		ii = i + Ny[m];
		jj = j + Nx[m];

		del = d_image[i * d_imagePitch + j] - d_image[ii * d_imagePitch + jj];

		C = sqrt(del*del);

		g = 1 - (C / maxC); //attack force

		//if (C > max)
		//	max = C;

		g *= d_strength[ii * d_strengthPitch + jj];

		if (g > d_strength[i * d_strengthPitch + j]) //attack succeeds
		{
			d_newStrength[i * d_strengthPitch + j] = g;
			d_newLabels[i * d_labelPitch + j] = d_labels[ii * d_labelPitch + jj];
			*d_converged = 0; // keep iterating
		}

	}

}

///////////////////////////////////////////////////////////////////////////////////////////
//Checker board approach 
///////////////////////////////////////////////////////////////////////////////////////////
static __global__ void d_growcuts_Checker(int *d_labels,  size_t d_labelPitch, float *d_strength, size_t d_strengthPitch,
	float *d_image, size_t d_imagePitch, int *d_converged, int ncols, int nrows, int d)

{

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int bOdd;

	if (y & 1)
		bOdd = 1; // row i is odd
	else
		bOdd = 0;

	//flip logic
	if (d == 1)
		bOdd = !bOdd;

	//center of block
	const int j = x * 2.0f + bOdd;
	const int i = y;

	if (j < 1 || i < 1 || j >= ncols - 1 || i >= nrows - 1)
		return;

	/*if (d_labels[i * d_labelPitch + j] == 999 || d_labels[i* d_labelPitch + j] == -999)
		return;*/

	if (d_strength[i * d_labelPitch + j] == 1.0f)
		return;

	float del, g, C;
	float maxC = 7000.0f;
	int ii, jj;
	int Nx[] = { -1, 1, 0, 0 }; //4-neighbors
	int Ny[] = { 0, 0, -1, 1 };
	int nNeighbors = 4;

	//for every neighbor q
	for (int m = 0; m < nNeighbors; m++)
	{

		ii = i + Ny[m];
		jj = j + Nx[m];

		del = d_image[i * d_imagePitch + j] - d_image[ii * d_imagePitch + jj];

		C = sqrt(del*del);

		g = 1 - (C / maxC); //attack force

		//if (C > max)
		//	max = C;

		g *= d_strength[ii * d_strengthPitch + jj];

		if (g > d_strength[i * d_strengthPitch + j]) //attack succeeds
		{
			d_strength[i * d_strengthPitch + j] = g;
			d_labels[i * d_labelPitch + j] = d_labels[ii * d_labelPitch + jj];
			*d_converged = 0; // keep iterating
		}

	}

}